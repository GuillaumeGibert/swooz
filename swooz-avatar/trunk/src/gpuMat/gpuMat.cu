#include "hip/hip_runtime.h"

#include <cutil.h>
#include "cula.h"
#include <iostream>

#include "configCuda.h"




//using namespace swUtil;
void cudaDummyCall(int argc, char **argv)
{
    // init CUDA
    //CUT_DEVICE_INIT(argc, argv);
    hipFree(0);
}

int inverseMatSgesv(float *aFInputMat, float *aFOutputInvMat, int i32SizeSquareMat)
{
    culaStatus l_oStatus;

    // init cula
    l_oStatus = culaInitialize();
    // check error
    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    int *l_aI32Ipiv = new int[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

    // launch gpu computing
    l_oStatus = culaSgesv(i32SizeSquareMat, i32SizeSquareMat, aFInputMat, i32SizeSquareMat, l_aI32Ipiv, aFOutputInvMat, i32SizeSquareMat);

    delete[] l_aI32Ipiv;

    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    culaShutdown();

    return 0;
}

//// Matrix multiplication kernel called by MatMul()
//__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
//{
//    // Each thread computes one element of C
//    // by accumulating results into Cvalue
//    float Cvalue = 0.f;
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int col = blockIdx.x * blockDim.x + threadIdx.x;

//    if(row > A.height || col > B.width)
//        return;

//    for (int e = 0; e < A.width; ++e)
//        Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
//    C.elements[row * C.width + col] = Cvalue;
//}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
//void matMult(const Matrix A, const Matrix B, Matrix C)
//{
//    // Load A and B to device memory
//    Matrix d_A;
//    d_A.width = A.width;
//    d_A.height = A.height;
//    size_t size = A.width * A.height * sizeof(float);
//    hipError_t err = hipMalloc(&d_A.elements, size);
////    printf("CUDA malloc A: %s\n",hipGetErrorString(err));
//    err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
////    printf("Copy A to device: %s\n",hipGetErrorString(err));

//    Matrix d_B;
//    d_B.width = B.width;
//    d_B.height = B.height;
//    size = B.width * B.height * sizeof(float);
//    err = hipMalloc(&d_B.elements, size);
////    printf("CUDA malloc B: %s\n",hipGetErrorString(err));
//    err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
////    printf("Copy B to device: %s\n",hipGetErrorString(err));

//    // Allocate C in device memory
//    Matrix d_C;
//    d_C.width = C.width;
//    d_C.height = C.height;
//    size = C.width * C.height * sizeof(float);
//    err = hipMalloc(&d_C.elements, size);
////    printf("CUDA malloc C: %s\n",hipGetErrorString(err));

//    // Invoke kernel
//    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
//    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x,
//           (A.height + dimBlock.y - 1) / dimBlock.y);

//    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
//    err = hipDeviceSynchronize();
////    printf("Run kernel: %s \n", hipGetErrorString(err));

//    // Read C from device memory
//    err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
////    printf("Copy C off of device: %s\n",hipGetErrorString(err));

//    // Free device memory
//    hipFree(d_A.elements);
//    hipFree(d_B.elements);
//    hipFree(d_C.elements);
//}


// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void transpose_kernel(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCKSIZE][BLOCKSIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCKSIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCKSIZE + threadIdx.y;
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCKSIZE + threadIdx.x;
    yIndex = blockIdx.x * BLOCKSIZE + threadIdx.y;
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}


// This naive transpose kernel suffers from completely non-coalesced writes.
// It can be up to 10x slower than the kernel above for large matrices.
__global__ void transpose_naive_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = idata[index_in];
   }
}

void transpose(float *idata, float *odata, int width, int height)
{
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * width * height;

    // allocate device memory
    float* d_idata;
    float* d_odata;
    hipMalloc( (void**) &d_idata, mem_size);
    hipMalloc( (void**) &d_odata, mem_size);

    // copy host memory to device
    hipMemcpy( d_idata, idata, mem_size,
                                hipMemcpyHostToDevice);

    // setup execution parameters
    dim3 grid(width / BLOCKSIZE, height / BLOCKSIZE, 1);
    dim3 threads(BLOCKSIZE, BLOCKSIZE, 1);

    // warmup so we don't time CUDA startup
//    transpose_naive_kernel<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);

    // synchronize here, so we make sure that we don't count any time from the asynchronize kernel launches.
    hipDeviceSynchronize();

    printf("Transposing a %d by %d matrix of floats...\n", width, height);

    // execute the kernel
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);
    hipDeviceSynchronize();

    // copy result from device to    host
    odata = (float*) malloc(mem_size);
    hipMemcpy( odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost);

    hipFree(d_idata);
    hipFree(d_odata);

    hipDeviceReset();
}





// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCKSIZE;
    Asub.height   = BLOCKSIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCKSIZE * row
                                         + BLOCKSIZE * col];
    return Asub;
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCKSIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCKSIZE][BLOCKSIZE];
        __shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCKSIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

 // Matrix multiplication - Host code
 // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
 void matMult(const Matrix A, const Matrix B, Matrix C, const int blockSize)
 {
     // Load A and B to device memory
     Matrix d_A;
     d_A.width = d_A.stride = A.width; d_A.height = A.height;
     size_t size = A.width * A.height * sizeof(float);
     hipMalloc(&d_A.elements, size);
     hipMemcpy(d_A.elements, A.elements, size,
                hipMemcpyHostToDevice);
     Matrix d_B;
     d_B.width = d_B.stride = B.width; d_B.height = B.height;
     size = B.width * B.height * sizeof(float);
     hipMalloc(&d_B.elements, size);
     hipMemcpy(d_B.elements, B.elements, size,
     hipMemcpyHostToDevice);

     // Allocate C in device memory
     Matrix d_C;
     d_C.width = d_C.stride = C.width; d_C.height = C.height;
     size = C.width * C.height * sizeof(float);
     hipMalloc(&d_C.elements, size);

     // Invoke kernel
     dim3 dimBlock(blockSize, blockSize);
     dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
     MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

     // Read C from device memory
     hipMemcpy(C.elements, d_C.elements, size,
                hipMemcpyDeviceToHost);

     // Free device memory
     hipFree(d_A.elements);
     hipFree(d_B.elements);
     hipFree(d_C.elements);
 }








// //unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
// //unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

// __global__ void GetVals(sparse_matrix A, sparse_matrix B,sparse_matrix C, int* indexTable)
// {
//     const int laneId = threadIdx.x;
//     const int blockId = blockIdx.x;
//     __shared__ unsigned int back;
//     int rowAStart; // The index into A.jc and A.val
//     int rowAEnd; // The boundary index for A
//     float valA; // The value of the current A nonzero
//     int rowBStart; // The index into B.jc and B.val
//     int rowBEnd; // The boundary index for B
//     int colB; // The current column in B being used
//     int rowCStart; // The index into C.jc and C.val
//     int rowCEnd; // The boundary index for C
//     int hash; // The calculated hash value
//     int i, j; // Loop iterators
//     // Set the global hash table to point to the space
//     // used by this warp
//     int* gColHashTable;
//     float* gValHashTable;
//     int globalEntries;
//     indexTable = &indexTable[C.cols * blockId];
//     if(laneId == 0)
//         back = 0;


//     for(int rowA = blockId; rowA < A.rows; rowA += gridDim.x)
//     {
//         rowAStart = A.ir[rowA];
//         rowAEnd = A.ir[rowA + 1];

//         for(i = laneId; i < C.cols; ++i)
//         {
//             indexTable[i] = -1;
//         }

//         __syncthreads();

//         // Set the location of the global hash table
//         rowCStart = C.ir[rowA];
//         rowCEnd = C.ir[rowA + 1];
//         globalEntries = rowCEnd - rowCStart;
//         gColHashTable = &C.jc[rowCStart];
//         gValHashTable = &C.val[rowCStart];

//         for(i = rowAStart; i < rowAEnd; ++i)
//         {
//             valA = A.val[i];
//             rowBStart = B.ir[A.jc[i]];
//             rowBEnd = B.ir[A.jc[i] + 1];
//             int curIdx;
//             int* storeInt;
//             float* storeFloat;
//             float valB;

//             for(j = rowBStart + laneId; __any(j < rowBEnd); j += warpSize)
//             {
//                 colB = j < rowBEnd ? B.jc[j] : -1;
//                 curIdx = colB == -1 ? -1 : indexTable[colB];
//                 hash = colB != -1 && curIdx == -1 ?
//                 atomicInc(&back, globalEntries - 1) : curIdx;
//                 storeInt = hash == -1 ? &hash : &indexTable[colB];
//                 *storeInt = hash;

//                 storeInt = hash == -1 ? &colB : &gColHashTable[hash];
//                 *storeInt = colB;
//                 valB = colB == -1 ? 1 : B.val[j];
//                 storeFloat = hash == -1 ? &valA : &gValHashTable[hash];
//                 *storeFloat += valB * valA;
//             }
//         } // For each nonzero in the A row
//     } // For each assigned row in A
// }

// __global__ void SortCols(sparse_matrix C, int maxRowNNZ, int* workQueue)
// {
//     const int laneId = threadIdx.x;

//     const int blockId = blockIdx.x;
//     // Dynamic shared memory
//     extern __shared__ int sharedMem[];
//     // The maximum size of the queue
//     const int queueSize = (maxRowNNZ / 2) + 1;
//     // The maximum number of passes needed
//     int maxShift = __log2f(C.cols) / RADIX_BITS;
//     // The number of passes for the work in the queue
//     int* workPasses = &workQueue[blockId * queueSize];
//     // The front of the bucket for the work in the queue
//     int* workFronts = &workQueue[gridDim.x * queueSize];
//     workFronts = &workFronts[blockId * queueSize];
//     // The back of the bucket for the work in the queue
//     int* workBacks = &workQueue[gridDim.x * queueSize * 2];
//     workBacks = &workBacks[blockId * queueSize];
//     int front; // The front of the work queue.
//     __shared__ unsigned int back; // The back of the work queue.
//     // Holds the sizes for the buckets being sorted by the threads
//     int* bucketSizes = &sharedMem[laneId * RADIX_BASE];
//     // The ending index of the buckets being sorted
//     int* bucketBounds = &sharedMem[blockDim.x * RADIX_BASE];
//     bucketBounds = &bucketBounds[laneId * RADIX_BASE];
//     int pass; // The pass number of the current bucket
//     int bucketFront; // The index of the front of the bucket
//     int bucketBack; // The index of the back of the bucket
//     int bucketIdx; // The index of an item in the bucket
//     int shiftCount; // The number of bits to shift to get the index
//     int iTmp; // A temporary variable for swapping
//     float fTmp;
//     int swapIdx; // The index to swap with
//     int queueIdx; // An index into the work queue
//     int prev; // The previous bucket offset
//     int subIdx;
//     for(int rowC = blockId; rowC < C.rows; rowC += gridDim.x)
//     {
//         // Skip if there are not non-zeros to sort
//         if(C.ir[rowC] == C.ir[rowC + 1])
//             continue;

//         // Clear the work queue
//         for(int i = laneId + 1; i < queueSize; i += blockDim.x)
//         {
//             workPasses[i] = -1;
//         }
//         workPasses[0] = 0;
//         workFronts[0] = C.ir[rowC];
//         workBacks[0] = C.ir[rowC + 1];
//         front = 0;
//         back = 1;
//         __syncthreads();

//         // While there is more work in the queue
//         while(front != back)
//         {
//             queueIdx = (front + laneId) % queueSize;
//             // Get the work
//             pass = workPasses[queueIdx];
//             bucketFront = workFronts[queueIdx];
//             bucketBack = workBacks[queueIdx];
//             // Clear this work
//             workPasses[queueIdx] = -1;
//             // Move the front forward
//             if((back > front && back - front <= blockDim.x) || (back < front && (back + queueSize) - front <= blockDim.x))
//             {
//                 front = back;
//             }
//             else
//             {
//                 front = (front + blockDim.x) % queueSize;
//             }

//             // There is work to do
//             if(pass >= 0)
//             {
//                 // Clear the bucket sizes
//                 for(int i = 0; i < RADIX_BASE; ++i)
//                 {
//                     bucketSizes[i] = 0;
//                 }
//                 shiftCount = (maxShift - pass) * RADIX_BITS;

//                 // First, determine the size of the buckets
//                 for(int i = bucketFront; i < bucketBack; ++i)
//                 {
//                     ++bucketSizes[(C.jc[i] >> shiftCount) & RADIX_MASK];
//                 }

//                 // Determine the indexes of the buckets and put
//                 // them into the work queue
//                 prev = bucketFront;
//                 for(int i = 0; i < RADIX_BASE; ++i)
//                 {
//                     // Determine the bucket end
//                     bucketIdx = bucketSizes[i] + prev;
//                     // Place the bucket into the work queue only
//                     // if it has items to be sorted
//                     if(bucketSizes[i] > 1)
//                     {
//                         queueIdx = atomicInc(&back, queueSize - 1);
//                         workPasses[queueIdx] = pass + 1;
//                         workFronts[queueIdx] = prev;
//                         workBacks[queueIdx] = bucketIdx;
//                     }
//                     // Store the bucket end
//                     bucketSizes[i] = bucketIdx;
//                     bucketBounds[i] = bucketIdx;
//                     prev = bucketIdx;
//                 }

//                 // Place the items into the buckets
//                 bucketIdx = bucketFront;
//                 while(bucketIdx != bucketBack)
//                 {
//                     subIdx = (C.jc[bucketIdx] >> shiftCount) & RADIX_MASK;
//                     swapIdx = --bucketSizes[subIdx];;
//                     // Done sorting this bucket, move to the next open one
//                     if(swapIdx == bucketIdx)
//                     {
//                         do
//                         {
//                             bucketIdx = bucketBounds[subIdx++];
//                         } while(bucketIdx != bucketBack && bucketSizes[subIdx] == bucketIdx);
//                     }
//                     else
//                     {
//                         // Swap swapIdx and bucketIdx
//                         iTmp = C.jc[swapIdx];
//                         C.jc[swapIdx] = C.jc[bucketIdx];
//                         C.jc[bucketIdx] = iTmp;
//                         fTmp = C.val[swapIdx];
//                         C.val[swapIdx] = C.val[bucketIdx];
//                         C.val[bucketIdx] = fTmp;
//                     }
//                 }
//             } // If this thread has work
//             __syncthreads();
//         } // While there is work to do
//     } // For all rows in C
// }

// //#include <thrust/version.h>
// //#include <cusp/version.h>

// //#include <cusp/multiply.h>
// //#include <cusp/array2d.h>
// //#include <cusp/print.h>

// //int testCuSparse(void)
// //{
// //    int cuda_major =  CUDA_VERSION / 1000;
// //    int cuda_minor = (CUDA_VERSION % 1000) / 10;

// //    int thrust_major = THRUST_MAJOR_VERSION;
// //    int thrust_minor = THRUST_MINOR_VERSION;

// //    int cusp_major = CUSP_MAJOR_VERSION;
// //    int cusp_minor = CUSP_MINOR_VERSION;

// //    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
// //    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
// //    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;


// //    // initialize matrix
// //    cusp::array2d<float, cusp::host_memory> A(2,2);
// //    A(0,0) = 10;  A(0,1) = 20;
// //    A(1,0) = 40;  A(1,1) = 50;

// //    // initialize input vector
// //    cusp::array1d<float, cusp::host_memory> x(2);
// //    x[0] = 1;
// //    x[1] = 2;

// //    // allocate output vector
// //    cusp::array1d<float, cusp::host_memory> y(2);

// //    // compute y = A * x
// //    cusp::multiply(A, x, y);

// //    // print y
// //    cusp::print(y);

// //    return 0;
// //}





// __global__ void GetNNZ(sparse_matrix A, sparse_matrix B, sparse_matrix C,int* workingSet)
// {
//     const int laneId = threadIdx.x;
//     const int warpId = blockIdx.x;
//     int* nonzeros;
//     int rowAStart, rowAEnd, rowBStart, rowBEnd;
//     int nnz;
//     int colC;
//     extern __shared__ int nzCount[];
//     nonzeros = &workingSet[warpId * B.cols];
//     // Iterate through each assigned row in A.
//     for(int rowA = warpId; rowA < A.rows; rowA += gridDim.x)
//     {
//         rowAStart = A.ir[rowA];
//         rowAEnd = A.ir[rowA + 1];
//         // There are no non-zeros in this row so continue
//         if(rowAStart == rowAEnd)
//         {
//             if (laneId == 0)
//                 C.ir[rowA] = 0;
//             __syncthreads();
//             continue;
//         }

//         // Reset the nz counts
//         nzCount[laneId] = 0;
//         // reset the nonzeros table
//         for (int i=laneId; i<B.cols; i+= warpSize)
//         {
//             nonzeros[i] = 0;
//         }
//         __syncthreads();

//         for(int i = rowAStart; i < rowAEnd; ++i)
//         {
//             rowBStart = B.ir[A.jc[i]];
//             rowBEnd = B.ir[A.jc[i]+1];

//             for (int j = rowBStart + laneId; j < rowBEnd; j += warpSize)
//             {
//                 colC = B.jc[j];
//                 nzCount[laneId] += nonzeros[colC] == 0;
//                 nonzeros[colC] = 1;
//             }
//             __syncthreads();
//         }

//         if(laneId == 0)
//         {
//             nnz = nzCount[0];
//             for(int i = 1; i < warpSize; ++i)
//             {
//                 nnz += nzCount[i];
//             }

//             C.ir[rowA] = nnz;
//         }

//         __syncthreads();
//     }
// }
