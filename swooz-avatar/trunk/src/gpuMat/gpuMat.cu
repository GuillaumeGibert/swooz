#include "hip/hip_runtime.h"

//#include <cutil.h>
#include "hipblas.h"
#include "cula.h"
#include <iostream>

#include "gpuMat/configCuda.h"


int doCulaSgesv(float *aFInputMat, float *aFOutputInvMat, int i32SizeSquareMat)
{
    culaStatus l_oStatus;

    // init cula
    l_oStatus = culaInitialize();
    // check error
    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    int *l_aI32Ipiv = new int[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

    // launch gpu computing
    l_oStatus = culaSgesv(i32SizeSquareMat, i32SizeSquareMat, aFInputMat, i32SizeSquareMat, l_aI32Ipiv, aFOutputInvMat, i32SizeSquareMat);

    delete[] l_aI32Ipiv;

    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    culaShutdown();

    return 0;
}

int LUDecomposition(float *aFMat, int i32SizeSquareMat)
{
    culaStatus l_oStatus;

    // init cula
    l_oStatus = culaInitialize();
    // check error
    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    int *l_aI32Ipiv = new int[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

    // launch gpu computing
    l_oStatus = culaSgetrf(i32SizeSquareMat, i32SizeSquareMat, aFMat, i32SizeSquareMat, l_aI32Ipiv);

    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    delete[] l_aI32Ipiv;

    return 1;
}

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void transpose_kernel(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCKSIZE][BLOCKSIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCKSIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCKSIZE + threadIdx.y;
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCKSIZE + threadIdx.x;
    yIndex = blockIdx.x * BLOCKSIZE + threadIdx.y;
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}


// This naive transpose kernel suffers from completely non-coalesced writes.
// It can be up to 10x slower than the kernel above for large matrices.
__global__ void transpose_naive_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = idata[index_in];
   }
}

void transpose(float *idata, float *odata, int width, int height)
{
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * width * height;

    // allocate device memory
    float* d_idata;
    float* d_odata;
    hipMalloc( (void**) &d_idata, mem_size);
    hipMalloc( (void**) &d_odata, mem_size);

    // copy host memory to device
    hipMemcpy( d_idata, idata, mem_size,
                                hipMemcpyHostToDevice);

    // setup execution parameters
    dim3 grid(width / BLOCKSIZE, height / BLOCKSIZE, 1);
    dim3 threads(BLOCKSIZE, BLOCKSIZE, 1);

    // warmup so we don't time CUDA startup
//    transpose_naive_kernel<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);

    // synchronize here, so we make sure that we don't count any time from the asynchronize kernel launches.
    hipDeviceSynchronize();

    printf("Transposing a %d by %d matrix of floats...\n", width, height);

    // execute the kernel
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);
    hipDeviceSynchronize();

    // copy result from device to    host
    odata = (float*) malloc(mem_size);
    hipMemcpy( odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost);

    hipFree(d_idata);
    hipFree(d_odata);

    hipDeviceReset();
}



// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCKSIZE;
    Asub.height   = BLOCKSIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCKSIZE * row
                                         + BLOCKSIZE * col];
    return Asub;
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCKSIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCKSIZE][BLOCKSIZE];
        __shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCKSIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

 // Matrix multiplication - Host code
 // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
 void matMult(const Matrix A, const Matrix B, Matrix C, const int blockSize)
 {
     // Load A and B to device memory
     Matrix d_A;
     d_A.width = d_A.stride = A.width; d_A.height = A.height;
     size_t size = A.width * A.height * sizeof(float);
     hipMalloc(&d_A.elements, size);
     hipMemcpy(d_A.elements, A.elements, size,
                hipMemcpyHostToDevice);
     Matrix d_B;
     d_B.width = d_B.stride = B.width; d_B.height = B.height;
     size = B.width * B.height * sizeof(float);
     hipMalloc(&d_B.elements, size);
     hipMemcpy(d_B.elements, B.elements, size,
     hipMemcpyHostToDevice);

     // Allocate C in device memory
     Matrix d_C;
     d_C.width = d_C.stride = C.width; d_C.height = C.height;
     size = C.width * C.height * sizeof(float);
     hipMalloc(&d_C.elements, size);

     // Invoke kernel
     dim3 dimBlock(blockSize, blockSize);
     dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
     MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

     // Read C from device memory
     hipMemcpy(C.elements, d_C.elements, size,
                hipMemcpyDeviceToHost);

     // Free device memory
     hipFree(d_A.elements);
     hipFree(d_B.elements);
     hipFree(d_C.elements);
 }


  //***************************************************************************************
  //void DecomposeLU( int M, int N, int lda , float* A,
  //                    int* permute, float epsilon, InfoStat& stat)
  //
  // M         :     Num of rows of A
  // N         :     Num of column of A
  // A         :     Float Matrix of size M*N
  //                on the output contains the result of the LU decomposition
  //                The diagonal elements for L are not stored in A ( assuming they are all 1)
  //lda        :    Leading dim of A lda < std::max(1,M)
  //P          :      Permutation vector of size M
  //epsilon    :     Epsilon (used to test for singularities)
  //stat          :  return status
  // **************************************************************************************



//  void DecomposeLU(int M, int N, int lda , float* A, int* P, float epsilon)//, InfoStat& stat)
//  {
//       hipblasStatus_t cuStat;
//       //Preconditions
//       if ( M<=0 || N<=0 || lda < std::max(1,M) )
//       {
//           printf("bad preconditions\n");
// //           stat._info = -1;
// //           if (M<=0)
// //               stat._str = "M<=0";
// //           if (N<=0)
// //               stat._str = "M<=0";
// //           if (lda < std::max(1,M))
// //               stat._str = "lda < std::max(1,M)";
//            return;
//       }
//       int minDim = std::min( M, N );
//       for (int k=0; k<minDim-1; k++)
//       {
//            int pivotRow = k-1+hipblasIsamax(M-k,A+k + k*lda, 1); // row relative to the current submatrix
//            int kp1 = k+1;
//            P[k] = pivotRow;
//            if (pivotRow!=k)
//            {
//                 hipblasSswap(N, A+pivotRow, lda, A+k, lda);
//            }
//            float valcheck;
//            hipblasGetVector(1,sizeof(float),A+k+ k*lda, 1, &valcheck, 1);
//            if (fabs(valcheck) < epsilon)
//            {
//                printf("Matrix is Singular\n");
// //                stat._info =k+1;
// //                stat._str = " Matrix is Singular ";
//                 return;
//            }
//            if (kp1 < M)
//           {
//                hipblasSscal(M-kp1, 1.0f/valcheck,A+kp1+ k*lda, 1);
//           }
//           if ( kp1 < minDim )
//           {
//                hipblasSger (M-kp1, N-kp1, -1.0f,A+kp1+ k*lda, 1, A+k+ kp1*lda, lda,A+ kp1*lda+kp1, lda);
//           }
//       }
// //      CHECK_CUBLAS("decomposeLU pb");
//  }




//// DecomposeBlockedLU(A.size(), A.size(), std::max(1,A.size()),   );

// //***************************************************************************************
// //void DecomposeBlockedLU ( int M, int N,int lda,
// //                          float *A,
// //                          int* P, int blockSize,float epsilon, InfoStat &stat )
// //
// // M            :   Num of rows of A
// // N            :   Num of column of A
// // A            :   Float Matrix of size M*N
// //                  on the output contains the result of the LU decomposition
// //                  The diagonal elements for L are not stored in A ( assuming they are all 1)
// //lda           :   Leading dim of A lda < std::max(1,M)
// //P             :   Permutation vector of size M
// //blockSize     :   Size of the submatrices
// //                  if blockSize>=M || blockSize==1 unblocked decomposition is called
// //epsilon       :   Epsilon (used to test for singularities)
// //stat          :  return status
// // **************************************************************************************
// void DecomposeBlockedLU (   int M, int N,int lda,
//                             float *A,
//                             int* P, int blockSize,float epsilon)//, InfoStat &stat )
// {

//     hipblasStatus_t cuStat;
//     //Preconditions
//     if (M < 0 || N < 0 || lda < std::max(1,M) )
//     {
////         stat._info = -1;
////         if (M<=0)
////             stat._str = "M<=0";
////         if (N<=0)
////             stat._str = "M<=0";
////         if (lda < std::max(1,M))
////             stat._str = "lda < std::max(1,M)";

//         return;
//     }

//     int minSize = std::min(M,N);

//     if ( blockSize > minSize || blockSize == 1)
//     {
//         //straight LU decomposition
//         DecomposeLU( M, N, lda, A, P, epsilon);//, stat);
//     }
//     else
//     {
//         //blocked decomposition
//         for (int i =0; i< minSize ; i+=blockSize)
//         {
//             int realBlockSize  = std::min(minSize - i, blockSize);

//             //decompose the current rectangular block
//             DecomposeLU( M-i, realBlockSize, lda, A+i+i*lda, P+i, epsilon);//, stat);

//             //adjust pivot infos
//             //Todo : write a kernel for that
//             for (int p = i; p< std::min( M, i+realBlockSize)-1; p++)
//             {
//                     P[p] = P[p]+i;
//                     if (P[p] != p)
//                     {
//                         // Apply interchanges to columns 0:i.
//                         hipblasSswap(i, A+p , lda, A+ P[p], lda);
//                         // Apply interchanges to columns i+blockSize:N.
//                         hipblasSswap(N-i-realBlockSize, A+p+(i+realBlockSize)*lda , lda, A+ P[p]+(i+realBlockSize)*lda, lda);
//                     }

//             }

//             // Compute block row of U.
//             hipblasStrsm( 'l','l','n','u', realBlockSize, N-i-realBlockSize, 1.0f,
//                          A +i +i*lda, lda, A +i + (i+realBlockSize)*lda, lda);
////             CHECK_CUBLAS("decomposeBlockedLU hipblasStrsm");

//             if (i+realBlockSize < M)
//             {
//                  hipblasSgemm('n','n',  M-i-realBlockSize, N-i-realBlockSize, realBlockSize,
//                              -1.0f,
//                              A+i+realBlockSize+i*lda,lda,
//                              A+i+(realBlockSize+i)*lda,lda,
//                              1.0f,
//                              A+i+realBlockSize+(realBlockSize+i)*lda,lda );
////                  CHECK_CUBLAS("decomposeBlockedLU hipblasSgemm");
//             }
//         }
//     }

// }


// int decomp(float* src_d, float *dst_d, int n)
// {
//     hipblasHandle_t handle;
//     hipblasCreate(&handle);

//     int batchSize = 1;
//     int *P, *INFO;
//     hipMalloc<int>(&P,n * batchSize * sizeof(int));
//     hipMalloc<int>(&INFO,batchSize * sizeof(int));

//     int lda = n;//spitch/sizeof(float);

//     float *A[] = { src_d };
//     float** A_d;
//     hipMalloc<float*>(&A_d,sizeof(A));
//     hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice);

//     hipblasStatus_t status = hipblasSgetrfBatched(handle,
//                                                 n,
//                                                 A_d,
//                                                 lda,
//                                                 P,
//                                                 INFO,
//                                                 batchSize);


//      int INFOh = 0;
//      hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost);

//      if(INFOh == n)
//      {
//              /* Singular */
//              hipblasDestroy(handle);
//              hipFree(INFO);
//              hipFree(P);
//              hipFree(A_d);
//              return -1;
//      }

//      if(HIPBLAS_STATUS_SUCCESS != status)
//      {
//              hipblasDestroy(handle);
//              hipFree(INFO);
//              hipFree(P);
//              hipFree(A_d);
//              return -2;
//      }
// }


//// void _decomposeLU2(float *src_d, float *dst_d, int n)
// void _decomposeLU2(float *src_d, int n)
// {
//     hipError_t cudaStat;
//     hipblasStatus_t stat;
//     hipblasHandle_t handle;
////     hipblasCreate(&handle );

//     int batchSize = n;
//     int *P, *INFO;
//     cudaStat = hipMalloc<int>(&P,n * batchSize * sizeof(int));
//     printf("malloc1 : %d\n", cudaStat);
//     cudaStat = hipMalloc<int>(&INFO, batchSize * sizeof(int));
//     printf("malloc2 : %d\n", cudaStat);

//     float *A[] = {src_d};
////     float *LU[] = {dst_d};

//     float** A_d;
//     cudaStat = hipMalloc<float*>(&A_d, sizeof(A));
//     printf("malloc3 : %d\n", cudaStat);

//     cudaStat =hipMemcpy(A_d, A, sizeof(A), hipMemcpyHostToDevice);
//     printf("memcopy : %d\n", cudaStat);

//     hipblasStatus_t statut =
//             hipblasSgetrfBatched(handle,    // handle to the CUBLAS library context.
//                                 n,         // number of rows and columns of Aarray[i].
//                                 A_d,       // array of pointers to <type> array, with each array of dim. n x n with lda>=max(1,n).
//                                 n,         // leading dimension of two-dimensional array used to store each matrix Aarray[i].
//                                 P,         // array of size n x batchSize that contains the pivoting sequence of each factorization of Aarray[i] stored in a linear fashion.
//                                 INFO,      // array of size batchSize that info(=infoArray[i]) contains the information of factorization of Aarray[i].
//                                 batchSize  // number of pointers contained in A
//                                );
//     printf("statut : %d\n", statut);

//     int INFOh = 0;
//     hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost);
//     printf("Info : %d\n", INFOh);

////     hipMemcpy(LU, A_d, sizeof(A), hipMemcpyHostToDevice);
//     hipMemcpy(A, A_d, sizeof(A), hipMemcpyHostToDevice);

//     hipFree(A_d);
//     hipFree(P);
//     hipFree(INFO);


////     hipblasDestroy(handle);
// }



// void _decomposeLU(float **A, int rows)
// {
//     int *P__ = new int[rows*rows];
//     int *infoArray = new int[rows];


////     float** A_d;
////    hipMalloc<float*>(&A_d,sizeof(A));

//     hipblasHandle_t handle;
//     hipblasStatus_t statut = hipblasSgetrfBatched(handle,   // handle to the CUBLAS library context.
//                                                 rows,  // number of rows and columns of Aarray[i].
//                                                 A,     // array of pointers to <type> array, with each array of dim. n x n with lda>=max(1,n).
//                                                 rows,        // leading dimension of two-dimensional array used to store each matrix Aarray[i].
//                                                 P__,      //array of size n x batchSize that contains the pivoting sequence of each factorization of Aarray[i] stored in a linear fashion.
//                                                 infoArray,//array of size batchSize that info(=infoArray[i]) contains the information of factorization of Aarray[i].
//                                                 rows   //number of pointers contained in A
//                                                 );
//     printf("statut : %d\n", statut);

//     for(int ii = 0; ii < rows; ++ii)
////         if(infoArray[ii] != 0)
//             printf(" e[%d %d] ", ii, infoArray[ii]);

//     delete[] P__;
//     delete[] infoArray;
// }





// void testCublas()
// {
////     float *data;

////     DecomposeLU(A.size(), A.size(), std::max(1,A.size()), A.data(), p.datat(), 0.f)

//////     float **data = new float[];


////     //     hipblasStatus_t hipblasSgetrfBatched(hipblasHandle_t handle,   handle to the CUBLAS library context.
////     //                                        int n,                   number of rows and columns of Aarray[i].
////     //                                        float *Aarray[],         array of pointers to <type> array, with each array of dim. n x n with lda>=max(1,n).
////     //                                        int lda,                 leading dimension of two-dimensional array used to store each matrix Aarray[i].
////     //                                        int *PivotArray,         array of size n x batchSize that contains the pivoting sequence of each factorization of Aarray[i] stored in a linear fashion.
////     //                                        int *infoArray,          array of size batchSize that info(=infoArray[i]) contains the information of factorization of Aarray[i].
////     //                                        int batchSize);          number of pointers contained in A

////     // LU Factorization
////     int *pivot;
////     int *info;
////     float **testF;
////     hipblasHandle_t handle;
////     hipblasStatus_t statu = hipblasSgetrfBatched(
////                                                handle,
////                                                100,
////                                                testF,
////                                                0,
////                                                pivot,
////                                                info,
////                                                10
////                                                );

// }




// //unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
// //unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

// __global__ void GetVals(sparse_matrix A, sparse_matrix B,sparse_matrix C, int* indexTable)
// {
//     const int laneId = threadIdx.x;
//     const int blockId = blockIdx.x;
//     __shared__ unsigned int back;
//     int rowAStart; // The index into A.jc and A.val
//     int rowAEnd; // The boundary index for A
//     float valA; // The value of the current A nonzero
//     int rowBStart; // The index into B.jc and B.val
//     int rowBEnd; // The boundary index for B
//     int colB; // The current column in B being used
//     int rowCStart; // The index into C.jc and C.val
//     int rowCEnd; // The boundary index for C
//     int hash; // The calculated hash value
//     int i, j; // Loop iterators
//     // Set the global hash table to point to the space
//     // used by this warp
//     int* gColHashTable;
//     float* gValHashTable;
//     int globalEntries;
//     indexTable = &indexTable[C.cols * blockId];
//     if(laneId == 0)
//         back = 0;


//     for(int rowA = blockId; rowA < A.rows; rowA += gridDim.x)
//     {
//         rowAStart = A.ir[rowA];
//         rowAEnd = A.ir[rowA + 1];

//         for(i = laneId; i < C.cols; ++i)
//         {
//             indexTable[i] = -1;
//         }

//         __syncthreads();

//         // Set the location of the global hash table
//         rowCStart = C.ir[rowA];
//         rowCEnd = C.ir[rowA + 1];
//         globalEntries = rowCEnd - rowCStart;
//         gColHashTable = &C.jc[rowCStart];
//         gValHashTable = &C.val[rowCStart];

//         for(i = rowAStart; i < rowAEnd; ++i)
//         {
//             valA = A.val[i];
//             rowBStart = B.ir[A.jc[i]];
//             rowBEnd = B.ir[A.jc[i] + 1];
//             int curIdx;
//             int* storeInt;
//             float* storeFloat;
//             float valB;

//             for(j = rowBStart + laneId; __any(j < rowBEnd); j += warpSize)
//             {
//                 colB = j < rowBEnd ? B.jc[j] : -1;
//                 curIdx = colB == -1 ? -1 : indexTable[colB];
//                 hash = colB != -1 && curIdx == -1 ?
//                 atomicInc(&back, globalEntries - 1) : curIdx;
//                 storeInt = hash == -1 ? &hash : &indexTable[colB];
//                 *storeInt = hash;

//                 storeInt = hash == -1 ? &colB : &gColHashTable[hash];
//                 *storeInt = colB;
//                 valB = colB == -1 ? 1 : B.val[j];
//                 storeFloat = hash == -1 ? &valA : &gValHashTable[hash];
//                 *storeFloat += valB * valA;
//             }
//         } // For each nonzero in the A row
//     } // For each assigned row in A
// }

// __global__ void SortCols(sparse_matrix C, int maxRowNNZ, int* workQueue)
// {
//     const int laneId = threadIdx.x;

//     const int blockId = blockIdx.x;
//     // Dynamic shared memory
//     extern __shared__ int sharedMem[];
//     // The maximum size of the queue
//     const int queueSize = (maxRowNNZ / 2) + 1;
//     // The maximum number of passes needed
//     int maxShift = __log2f(C.cols) / RADIX_BITS;
//     // The number of passes for the work in the queue
//     int* workPasses = &workQueue[blockId * queueSize];
//     // The front of the bucket for the work in the queue
//     int* workFronts = &workQueue[gridDim.x * queueSize];
//     workFronts = &workFronts[blockId * queueSize];
//     // The back of the bucket for the work in the queue
//     int* workBacks = &workQueue[gridDim.x * queueSize * 2];
//     workBacks = &workBacks[blockId * queueSize];
//     int front; // The front of the work queue.
//     __shared__ unsigned int back; // The back of the work queue.
//     // Holds the sizes for the buckets being sorted by the threads
//     int* bucketSizes = &sharedMem[laneId * RADIX_BASE];
//     // The ending index of the buckets being sorted
//     int* bucketBounds = &sharedMem[blockDim.x * RADIX_BASE];
//     bucketBounds = &bucketBounds[laneId * RADIX_BASE];
//     int pass; // The pass number of the current bucket
//     int bucketFront; // The index of the front of the bucket
//     int bucketBack; // The index of the back of the bucket
//     int bucketIdx; // The index of an item in the bucket
//     int shiftCount; // The number of bits to shift to get the index
//     int iTmp; // A temporary variable for swapping
//     float fTmp;
//     int swapIdx; // The index to swap with
//     int queueIdx; // An index into the work queue
//     int prev; // The previous bucket offset
//     int subIdx;
//     for(int rowC = blockId; rowC < C.rows; rowC += gridDim.x)
//     {
//         // Skip if there are not non-zeros to sort
//         if(C.ir[rowC] == C.ir[rowC + 1])
//             continue;

//         // Clear the work queue
//         for(int i = laneId + 1; i < queueSize; i += blockDim.x)
//         {
//             workPasses[i] = -1;
//         }
//         workPasses[0] = 0;
//         workFronts[0] = C.ir[rowC];
//         workBacks[0] = C.ir[rowC + 1];
//         front = 0;
//         back = 1;
//         __syncthreads();

//         // While there is more work in the queue
//         while(front != back)
//         {
//             queueIdx = (front + laneId) % queueSize;
//             // Get the work
//             pass = workPasses[queueIdx];
//             bucketFront = workFronts[queueIdx];
//             bucketBack = workBacks[queueIdx];
//             // Clear this work
//             workPasses[queueIdx] = -1;
//             // Move the front forward
//             if((back > front && back - front <= blockDim.x) || (back < front && (back + queueSize) - front <= blockDim.x))
//             {
//                 front = back;
//             }
//             else
//             {
//                 front = (front + blockDim.x) % queueSize;
//             }

//             // There is work to do
//             if(pass >= 0)
//             {
//                 // Clear the bucket sizes
//                 for(int i = 0; i < RADIX_BASE; ++i)
//                 {
//                     bucketSizes[i] = 0;
//                 }
//                 shiftCount = (maxShift - pass) * RADIX_BITS;

//                 // First, determine the size of the buckets
//                 for(int i = bucketFront; i < bucketBack; ++i)
//                 {
//                     ++bucketSizes[(C.jc[i] >> shiftCount) & RADIX_MASK];
//                 }

//                 // Determine the indexes of the buckets and put
//                 // them into the work queue
//                 prev = bucketFront;
//                 for(int i = 0; i < RADIX_BASE; ++i)
//                 {
//                     // Determine the bucket end
//                     bucketIdx = bucketSizes[i] + prev;
//                     // Place the bucket into the work queue only
//                     // if it has items to be sorted
//                     if(bucketSizes[i] > 1)
//                     {
//                         queueIdx = atomicInc(&back, queueSize - 1);
//                         workPasses[queueIdx] = pass + 1;
//                         workFronts[queueIdx] = prev;
//                         workBacks[queueIdx] = bucketIdx;
//                     }
//                     // Store the bucket end
//                     bucketSizes[i] = bucketIdx;
//                     bucketBounds[i] = bucketIdx;
//                     prev = bucketIdx;
//                 }

//                 // Place the items into the buckets
//                 bucketIdx = bucketFront;
//                 while(bucketIdx != bucketBack)
//                 {
//                     subIdx = (C.jc[bucketIdx] >> shiftCount) & RADIX_MASK;
//                     swapIdx = --bucketSizes[subIdx];;
//                     // Done sorting this bucket, move to the next open one
//                     if(swapIdx == bucketIdx)
//                     {
//                         do
//                         {
//                             bucketIdx = bucketBounds[subIdx++];
//                         } while(bucketIdx != bucketBack && bucketSizes[subIdx] == bucketIdx);
//                     }
//                     else
//                     {
//                         // Swap swapIdx and bucketIdx
//                         iTmp = C.jc[swapIdx];
//                         C.jc[swapIdx] = C.jc[bucketIdx];
//                         C.jc[bucketIdx] = iTmp;
//                         fTmp = C.val[swapIdx];
//                         C.val[swapIdx] = C.val[bucketIdx];
//                         C.val[bucketIdx] = fTmp;
//                     }
//                 }
//             } // If this thread has work
//             __syncthreads();
//         } // While there is work to do
//     } // For all rows in C
// }

// //#include <thrust/version.h>
// //#include <cusp/version.h>

// //#include <cusp/multiply.h>
// //#include <cusp/array2d.h>
// //#include <cusp/print.h>

// //int testCuSparse(void)
// //{
// //    int cuda_major =  CUDA_VERSION / 1000;
// //    int cuda_minor = (CUDA_VERSION % 1000) / 10;

// //    int thrust_major = THRUST_MAJOR_VERSION;
// //    int thrust_minor = THRUST_MINOR_VERSION;

// //    int cusp_major = CUSP_MAJOR_VERSION;
// //    int cusp_minor = CUSP_MINOR_VERSION;

// //    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
// //    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
// //    std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;


// //    // initialize matrix
// //    cusp::array2d<float, cusp::host_memory> A(2,2);
// //    A(0,0) = 10;  A(0,1) = 20;
// //    A(1,0) = 40;  A(1,1) = 50;

// //    // initialize input vector
// //    cusp::array1d<float, cusp::host_memory> x(2);
// //    x[0] = 1;
// //    x[1] = 2;

// //    // allocate output vector
// //    cusp::array1d<float, cusp::host_memory> y(2);

// //    // compute y = A * x
// //    cusp::multiply(A, x, y);

// //    // print y
// //    cusp::print(y);

// //    return 0;
// //}





// __global__ void GetNNZ(sparse_matrix A, sparse_matrix B, sparse_matrix C,int* workingSet)
// {
//     const int laneId = threadIdx.x;
//     const int warpId = blockIdx.x;
//     int* nonzeros;
//     int rowAStart, rowAEnd, rowBStart, rowBEnd;
//     int nnz;
//     int colC;
//     extern __shared__ int nzCount[];
//     nonzeros = &workingSet[warpId * B.cols];
//     // Iterate through each assigned row in A.
//     for(int rowA = warpId; rowA < A.rows; rowA += gridDim.x)
//     {
//         rowAStart = A.ir[rowA];
//         rowAEnd = A.ir[rowA + 1];
//         // There are no non-zeros in this row so continue
//         if(rowAStart == rowAEnd)
//         {
//             if (laneId == 0)
//                 C.ir[rowA] = 0;
//             __syncthreads();
//             continue;
//         }

//         // Reset the nz counts
//         nzCount[laneId] = 0;
//         // reset the nonzeros table
//         for (int i=laneId; i<B.cols; i+= warpSize)
//         {
//             nonzeros[i] = 0;
//         }
//         __syncthreads();

//         for(int i = rowAStart; i < rowAEnd; ++i)
//         {
//             rowBStart = B.ir[A.jc[i]];
//             rowBEnd = B.ir[A.jc[i]+1];

//             for (int j = rowBStart + laneId; j < rowBEnd; j += warpSize)
//             {
//                 colC = B.jc[j];
//                 nzCount[laneId] += nonzeros[colC] == 0;
//                 nonzeros[colC] = 1;
//             }
//             __syncthreads();
//         }

//         if(laneId == 0)
//         {
//             nnz = nzCount[0];
//             for(int i = 1; i < warpSize; ++i)
//             {
//                 nnz += nzCount[i];
//             }

//             C.ir[rowA] = nnz;
//         }

//         __syncthreads();
//     }
// }




 //#include <hip/hip_runtime.h>
 //#include <hipblas.h>


 //using namespace swUtil;
 //void cudaDummyCall()
 //{
 //    // init CUDA
 //    //CUT_DEVICE_INIT(argc, argv);
 //    hipFree(0);
 //}
 //void checkStatus(culaStatus status)
 //{
 //    char buf[256];

 //    if(!status)
 //        return;

 //    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
 //    printf("%s\n", buf);

 //    culaShutdown();
 //    exit(EXIT_FAILURE);
 //}
 //int doCulaSgesv(float *aFInputMat, float *aFOutputInvMat, int i32N, int i32NRHS)
 //{
 //    culaStatus l_oStatus;

 //    // init cula
 //    l_oStatus = culaInitialize();
 //    // check error
 //    if(l_oStatus != culaNoError)
 //    {
 //        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
 //        return -1; // TODO : create a throw
 //    }

 //    int *l_aI32Ipiv = new int[i32N * sizeof(float)];

 //    // launch gpu computing
 //    l_oStatus = culaSgesv(i32N, i32NRHS, aFInputMat, i32N, l_aI32Ipiv, aFOutputInvMat, i32N);

 //    delete[] l_aI32Ipiv;

 //    if(l_oStatus != culaNoError)
 //    {
 //        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
 //        return -1; // TODO : create a throw
 //    }

 //    culaShutdown();

 //    return 0;
 //}

 //int solveAX_B(float *A, int i32SizeSquareMat)
 //{
 //    culaStatus l_oStatus;

 //    // init cula
 //    l_oStatus = culaInitialize();
 //    // check error
 //    if(l_oStatus != culaNoError)
 //    {
 //        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
 //        return -1; // TODO : create a throw
 //    }

 //    int *l_aI32Ipiv = new int[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

 //    // launch gpu computing
 //    l_oStatus = culaSgetrf(i32SizeSquareMat, i32SizeSquareMat, A, i32SizeSquareMat, l_aI32Ipiv);

 //    if(l_oStatus != culaNoError)
 //    {
 //        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
 //        return -1; // TODO : create a throw
 //    }

 //    float *LU = A;

 //    float *L = LU;
 //    float *U = new float[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

 //    for(int ii = 0; ii < i32SizeSquareMat; ++ii)
 //    {
 //        for(int jj = 0; jj < i32SizeSquareMat; ++jj)
 //        {
 //            if(ii < jj)
 //            {
 //                U[ii*i32SizeSquareMat + jj] = 0.f;
 //            }
 //            else if(jj > ii)
 //            {
 //                U[ii*i32SizeSquareMat + jj] = LU[ii*i32SizeSquareMat + jj];
 //                L[ii*i32SizeSquareMat + jj] = 0.f;
 //            }
 //            else
 //            {
 //                L[ii*i32SizeSquareMat + jj] = 1.f;
 //                U[ii*i32SizeSquareMat + jj] = LU[ii*i32SizeSquareMat + jj];
 //            }
 //        }
 //    }


 ////    l_oStatus = culaSgesv(i32SizeSquareMat, i32SizeSquareMat, aFInputMat, i32SizeSquareMat, l_aI32Ipiv, aFOutputInvMat, i32SizeSquareMat);



 //    delete[] l_aI32Ipiv;
 //}

 //culaSgesv





 //  void _decomposeLU3(float *src_d, int n)
 //  {
 //      hipError_t cudaStat;
 //      hipblasStatus_t stat;
 //      hipblasHandle_t handle;

 //      int batchSize = n;
 //      int *P, *INFO;
 //      cudaStat = hipMalloc<int>(&P,n * batchSize * sizeof(int));
 //      printf("malloc1 : %d\n", cudaStat);
 //      cudaStat = hipMalloc<int>(&INFO, batchSize * sizeof(int));
 //      printf("malloc2 : %d\n", cudaStat);

 //      float *A[] = {src_d};
 // //     float *LU[] = {dst_d};

 //      printf("A sizeof %d\n", sizeof(A));

 //      float** A_d;


 //      cudaStat = hipMalloc<float*>(&A_d, sizeof(A));
 //      printf("malloc3 : %d\n", cudaStat);

 //      cudaStat =hipMemcpy(A_d, A, sizeof(A), hipMemcpyHostToDevice);
 //      printf("memcopy : %d\n", cudaStat);

 //      hipblasStatus_t statut =
 //              hipblasSgetrfBatched(handle,    // handle to the CUBLAS library context.
 //                                  n,         // number of rows and columns of Aarray[i].
 //                                  A_d,       // array of pointers to <type> array, with each array of dim. n x n with lda>=max(1,n).
 //                                  n,         // leading dimension of two-dimensional array used to store each matrix Aarray[i].
 //                                  P,         // array of size n x batchSize that contains the pivoting sequence of each factorization of Aarray[i] stored in a linear fashion.
 //                                  INFO,      // array of size batchSize that info(=infoArray[i]) contains the information of factorization of Aarray[i].
 //                                  batchSize  // number of pointers contained in A
 //                                 );
 //      printf("statut : %d\n", statut);

 //      int INFOh = 0;
 //      hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost);
 //      printf("Info : %d\n", INFOh);

 // //     hipMemcpy(LU, A_d, sizeof(A), hipMemcpyHostToDevice);
 //      hipMemcpy(A, A_d, sizeof(A), hipMemcpyHostToDevice);

 //      hipFree(A_d);
 //      hipFree(P);
 //      hipFree(INFO);
 //  }
