#include "hip/hip_runtime.h"

#include <cutil.h>
#include "cula.h"
#include <iostream>

#include "configCuda.h"


//using namespace swUtil;
void cudaDummyCall(int argc, char **argv)
{
    // init CUDA
    //CUT_DEVICE_INIT(argc, argv);
    hipFree(0);
}

int inverseMatSgesv(float *aFInputMat, float *aFOutputInvMat, int i32SizeSquareMat)
{
    culaStatus l_oStatus;

    // init cula
    l_oStatus = culaInitialize();
    // check error
    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error cuda init : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    int *l_aI32Ipiv = new int[i32SizeSquareMat * i32SizeSquareMat * sizeof(float)];

    // launch gpu computing
    l_oStatus = culaSgesv(i32SizeSquareMat, i32SizeSquareMat, aFInputMat, i32SizeSquareMat, l_aI32Ipiv, aFOutputInvMat, i32SizeSquareMat);

    delete[] l_aI32Ipiv;

    if(l_oStatus != culaNoError)
    {
        std::cerr << "Error culaSgesv : " << culaGetErrorInfo() << std::endl;
        return -1; // TODO : create a throw
    }

    culaShutdown();

    return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0.f;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row > A.height || col > B.width)
        return;

    for (int e = 0; e < A.width; ++e)
        Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
    C.elements[row * C.width + col] = Cvalue;
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void matMult(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipError_t err = hipMalloc(&d_A.elements, size);
//    printf("CUDA malloc A: %s\n",hipGetErrorString(err));
    err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
//    printf("Copy A to device: %s\n",hipGetErrorString(err));

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    err = hipMalloc(&d_B.elements, size);
//    printf("CUDA malloc B: %s\n",hipGetErrorString(err));
    err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
//    printf("Copy B to device: %s\n",hipGetErrorString(err));

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    err = hipMalloc(&d_C.elements, size);
//    printf("CUDA malloc C: %s\n",hipGetErrorString(err));

    // Invoke kernel
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x,
           (A.height + dimBlock.y - 1) / dimBlock.y);

    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
//    printf("Run kernel: %s \n", hipGetErrorString(err));

    // Read C from device memory
    err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
//    printf("Copy C off of device: %s\n",hipGetErrorString(err));

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}


// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void transpose_kernel(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCKSIZE][BLOCKSIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCKSIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCKSIZE + threadIdx.y;
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCKSIZE + threadIdx.x;
    yIndex = blockIdx.x * BLOCKSIZE + threadIdx.y;
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}


// This naive transpose kernel suffers from completely non-coalesced writes.
// It can be up to 10x slower than the kernel above for large matrices.
__global__ void transpose_naive_kernel(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = idata[index_in];
   }
}

void transpose(float *idata, float *odata, int width, int height)
{
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * width * height;

    // allocate device memory
    float* d_idata;
    float* d_odata;
    hipMalloc( (void**) &d_idata, mem_size);
    hipMalloc( (void**) &d_odata, mem_size);

    // copy host memory to device
    hipMemcpy( d_idata, idata, mem_size,
                                hipMemcpyHostToDevice);

    // setup execution parameters
    dim3 grid(width / BLOCKSIZE, height / BLOCKSIZE, 1);
    dim3 threads(BLOCKSIZE, BLOCKSIZE, 1);

    // warmup so we don't time CUDA startup
//    transpose_naive_kernel<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);

    // synchronize here, so we make sure that we don't count any time from the asynchronize kernel launches.
    hipDeviceSynchronize();

    printf("Transposing a %d by %d matrix of floats...\n", width, height);

    // execute the kernel
    transpose_kernel<<< grid, threads >>>(d_odata, d_idata, width, height);
    hipDeviceSynchronize();

    // copy result from device to    host
    odata = (float*) malloc(mem_size);
    hipMemcpy( odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost);

    hipFree(d_idata);
    hipFree(d_odata);

    hipDeviceReset();
}

