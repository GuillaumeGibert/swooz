#include "hip/hip_runtime.h"
/*
  Copyright (c) 2010 Toru Tamaki

  Permission is hereby granted, free of charge, to any person
  obtaining a copy of this software and associated documentation
  files (the "Software"), to deal in the Software without
  restriction, including without limitation the rights to use,
  copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the
  Software is furnished to do so, subject to the following
  conditions:

  The above copyright notice and this permission notice shall be
  included in all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
  EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
  OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
  NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
  HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
  WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
  FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
  OTHER DEALINGS IN THE SOFTWARE.
*/

	// cublas getError
	// AllocFailed	3	
	// ArchMismatch	8	
	// ExecutionFailed	13	
	// InternalError	14	
	// InvalidValue	7	
	// MappingError	11	
	// NotInitialized	1	
	// Success	0

#include <iostream>
#include <algorithm>
#include <cstdio>

#include "emicp/cutil.h"
#include "emicp/3dregistration.h"

#include "hipblas.h"

// uncomment if you do not use the viewer.
#define NOVIEWER



using namespace std;


void initCuda()
{    
	// init CUDA
//    CUT_DEVICE_INIT(param.argc, param.argv);
    hipSetDevice(0);
    hipFree(0);
}


void cudaReset()
{
	// reset CUDA
	CUDA_SAFE_CALL( hipDeviceReset() );	
}


__global__ static void updateA( int rowsA, int colsA, int pitchA,
        const float* d_Xx, const float* d_Xy, const float* d_Xz,
        const float* d_Yx, const float* d_Yy, const float* d_Yz,
        const float* d_R,  const float* d_t,
              float* d_A,        float  sigma_p2 )
{
	int r =  blockIdx.x * blockDim.x + threadIdx.x;
	int c =  blockIdx.y * blockDim.y + threadIdx.y;

	// Shared memory
	__shared__ float XxShare[BLOCK_SIZE];
	__shared__ float XyShare[BLOCK_SIZE];
	__shared__ float XzShare[BLOCK_SIZE];
	__shared__ float YxShare[BLOCK_SIZE];
	__shared__ float YyShare[BLOCK_SIZE];
	__shared__ float YzShare[BLOCK_SIZE];
	__shared__ float RShare[9]; // BLOCK_SIZE >= 9 is assumed
	__shared__ float tShare[3]; // BLOCK_SIZE >= 3 is assumed

    if(threadIdx.y == 0)
    {
        if(threadIdx.x < 9) // 0 <= threadIdx.x &&  // threadIdx.x is unsigned int, so always positive
		{
            RShare[threadIdx.x] = d_R[threadIdx.x];

			if(threadIdx.x < 3)
            {
                tShare[threadIdx.x] = d_t[threadIdx.x];
            }
		}
    }

	if(r < rowsA && c < colsA) // check for only inside the matrix A
	{ 
		if(threadIdx.x == 0)
		{
			XxShare[threadIdx.y] = d_Xx[c];
			XyShare[threadIdx.y] = d_Xy[c];
			XzShare[threadIdx.y] = d_Xz[c];
		}
		if(threadIdx.y == 0)
		{
			YxShare[threadIdx.x] = d_Yx[r];
			YyShare[threadIdx.x] = d_Yy[r];
			YzShare[threadIdx.x] = d_Yz[r];
		}

		__syncthreads();

		#define Xx XxShare[threadIdx.y]
		#define Xy XyShare[threadIdx.y]
		#define Xz XzShare[threadIdx.y]
		#define Yx YxShare[threadIdx.x]
		#define Yy YyShare[threadIdx.x]
		#define Yz YzShare[threadIdx.x]
		#define R(i) RShare[i]
		#define t(i) tShare[i]

		// #define Euclid(a,b,c) ((a)*(a)+(b)*(b)+(c)*(c))
		//     float tmp =
		//       Euclid(Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0)),
		//              Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1)),
		//              Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2)) );
		    
		//     tmp = expf(-tmp/sigma_p^2)


		float tmpX = Xx - (R(0)*Yx + R(1)*Yy + R(2)*Yz + t(0));
		float tmpY = Xy - (R(3)*Yx + R(4)*Yy + R(5)*Yz + t(1));
		float tmpZ = Xz - (R(6)*Yx + R(7)*Yy + R(8)*Yz + t(2));

		__syncthreads();

		tmpX *= tmpX;
		tmpY *= tmpY;
		tmpZ *= tmpZ;

		tmpX += tmpY;
		tmpX += tmpZ;

		tmpX /= sigma_p2;
		tmpX = expf(-tmpX);

		//float *A = (float*)((char*)d_A + c * pitchMinBytes) + r;

		d_A[c * pitchA + r] = tmpX;
	}
}


__global__ static void normalizeRowsOfA(int rowsA, int colsA, int pitchA, float *d_A, const float *d_C)		 
{
  
	int r =  blockIdx.x * blockDim.x + threadIdx.x;
	int c =  blockIdx.y * blockDim.y + threadIdx.y;

	// Shared memory
	__shared__ float d_CShare[BLOCK_SIZE];


	if(r < rowsA && c < colsA)// check for only inside the matrix A
	{ 

		if(threadIdx.y == 0)
			d_CShare[threadIdx.x] = d_C[r];

		__syncthreads();

		if(d_CShare[threadIdx.x] > 10e-7f)
		// each element in A is normalized C, then squre-rooted
			d_A[c * pitchA + r] = sqrtf( d_A[c * pitchA + r] / d_CShare[threadIdx.x] );
		else
			d_A[c * pitchA + r] = 1.0f/colsA; // ad_hoc code to avoid 0 division

		__syncthreads();
	}
}

__global__ static void elementwiseDivision(int Xsize, float* d_Xx, float* d_Xy, float* d_Xz, const float* d_lambda)
{
	int x =  blockIdx.x * blockDim.x + threadIdx.x;

	if(x < Xsize) // check for only inside X
	{ 
		float l_lambda = d_lambda[x];
		d_Xx[x] /= l_lambda;
		d_Xy[x] /= l_lambda;
		d_Xz[x] /= l_lambda;
	}
}

__global__ static void elementwiseMultiplication( int Xsize,
						  float* d_Xx, float* d_Xy, float* d_Xz,
						  const float* d_lambda)
{
	int x =  blockIdx.x * blockDim.x + threadIdx.x;

	if(x < Xsize) // check for only inside X
	{ 
		float l_lambda = d_lambda[x];
		d_Xx[x] *= l_lambda;
		d_Xy[x] *= l_lambda;
		d_Xz[x] *= l_lambda;
	}
}


__global__ static void centeringXandY( int rowsA,
				       const float* d_Xc, const float* d_Yc,
				       const float* d_Xx, const float* d_Xy, const float* d_Xz,
				       const float* d_Yx, const float* d_Yy, const float* d_Yz,
				       float* d_XxCenterd, float* d_XyCenterd, float* d_XzCenterd,
				       float* d_YxCenterd, float* d_YyCenterd, float* d_YzCenterd )
{
	// do for both X and Y at the same time

	int r =  blockIdx.x * blockDim.x + threadIdx.x;

	// Shared memory
	__shared__ float Xc[3];
	__shared__ float Yc[3];

	if(threadIdx.x < 6) // assume blocksize >= 6
		if(threadIdx.x < 3) 
			Xc[threadIdx.x] = d_Xc[threadIdx.x];
		else
			Yc[threadIdx.x - 3] = d_Yc[threadIdx.x - 3];

	if(r < rowsA) // check for only inside the vectors
	{
		__syncthreads();

		d_XxCenterd[r] = d_Xx[r] - Xc[0];
		d_XyCenterd[r] = d_Xy[r] - Xc[1];
		d_XzCenterd[r] = d_Xz[r] - Xc[2];

		d_YxCenterd[r] = d_Yx[r] - Yc[0];
		d_YyCenterd[r] = d_Yy[r] - Yc[1];
		d_YzCenterd[r] = d_Yz[r] - Yc[2];

		__syncthreads();
	}
}



bool emicp(int Xsize, int Ysize,
	   const float* h_X,
	   const float* h_Y,
	   float* h_R, float* h_t, 
	   registration::registrationParameters param)	   
{    
	// initialize parameters
	bool l_bNoError    = true;
	float sigma_p2 	   = param.sigma_p2;
	float sigma_inf    = param.sigma_inf;
	float sigma_factor = param.sigma_factor;
    float d_02         = param.d_02;

	//
	// memory allocation
	//

	// example: memCUDA(Xx, Xsize);   // declare d_Xx. no copy.
	#define memCUDA(var,num)						\
		float* d_ ## var; CUDA_SAFE_CALL(hipMalloc((void**) &(d_ ## var), sizeof(float)*num));


	// example:   memHostToCUDA(Xx, Xsize);   // declera d_Xx, then copy h_Xx to d_Xx.
	#define memHostToCUDA(var,num)						\
		float* d_ ## var; CUDA_SAFE_CALL(hipMalloc((void**) &(d_ ## var), sizeof(float)*num)); \
		CUDA_SAFE_CALL(hipMemcpy(d_ ## var, h_ ## var, sizeof(float)*num, hipMemcpyHostToDevice));

	memHostToCUDA(X, Xsize*3);
	float* d_Xx = &d_X[Xsize*0];
	float* d_Xy = &d_X[Xsize*1];
	float* d_Xz = &d_X[Xsize*2];

	memHostToCUDA(Y, Ysize*3);
	float* d_Yx = &d_Y[Ysize*0];
	float* d_Yy = &d_Y[Ysize*1];
	float* d_Yz = &d_Y[Ysize*2];

	memCUDA(Xprime, Ysize*3);
	float *d_XprimeX = &d_Xprime[Ysize*0];
	float *d_XprimeY = &d_Xprime[Ysize*1];
	float *d_XprimeZ = &d_Xprime[Ysize*2];

	float *d_XprimeCenterd = d_Xprime;
	float *d_XprimeCenterdX = &d_XprimeCenterd[Ysize*0];
	float *d_XprimeCenterdY = &d_XprimeCenterd[Ysize*1];
	float *d_XprimeCenterdZ = &d_XprimeCenterd[Ysize*2];

	memCUDA(YCenterd, Ysize*3);
	float *d_YCenterdX = &d_YCenterd[Ysize*0];
	float *d_YCenterdY = &d_YCenterd[Ysize*1];
	float *d_YCenterdZ = &d_YCenterd[Ysize*2];

	// center of X, Y
	float h_Xc[3], h_Yc[3];
	memCUDA(Xc, 3);
	memCUDA(Yc, 3);

	// R, t
	memHostToCUDA(R, 3*3);
	memHostToCUDA(t, 3);

	// S for finding R, t
	float h_S[9];
	memCUDA(S, 9);


    // NOTESTASM on matrix A
	// number of rows:     Ysize, or rowsA
	// number of columns : Xsize, or colsA
	// 
	//                    [0th in X] [1st]  ... [(Xsize-1)] 
	// [0th point in Y] [ A(0,0)     A(0,1) ... A(0,Xsize-1)      ] 
	// [1st           ] [ A(1,0)     A(1,1) ...                   ]
	// ...              [ ...                                     ]
	// [(Ysize-1)     ] [ A(Ysize-1, 0)     ... A(Ysize-1,Xsize-1)]
	//
	// 
	// CAUTION on matrix A
	// A is allcoated as a column-maijor format for the use of cublas.
	// This means that you must acces an element at row r and column c as:
	// A(r,c) = A[c * pitchA + r]

	int rowsA = Ysize;
	int colsA = Xsize;

	// pitchA: leading dimension of A, which is ideally equal to rowsA,
	//          but actually larger than that.
	int pitchA = (rowsA / 4 + 1) * 4;

	memCUDA(A, pitchA*colsA);

	// a vector with all elements of 1.0f
	float* h_one = new float [max(Xsize,Ysize)];
	for(int t = 0; t < max(Xsize,Ysize); t++) h_one[t] = 1.0f;
	memHostToCUDA(one, max(Xsize,Ysize));


	memCUDA(sumOfMRow, rowsA);
	memCUDA(C, rowsA); // sum of a row in A
	memCUDA(lambda, rowsA); // weight of a row in A

	//
	// threads
	//


	// for 2D block
	dim3 dimBlockForA(BLOCK_SIZE, BLOCK_SIZE); // a block is (BLOCK_SIZE*BLOCK_SIZE) threads
	dim3 dimGridForA( (pitchA + dimBlockForA.x - 1) / dimBlockForA.x,
		    (colsA  + dimBlockForA.y - 1) / dimBlockForA.y);

	// for 1D block
	int threadsPerBlockForYsize = 512; // a block is 512 threads
	int blocksPerGridForYsize = (Ysize + threadsPerBlockForYsize - 1 ) / threadsPerBlockForYsize;

	//
	// timer
	//

//	#define START_TIMER(timer) \
//		if(!param.notimer){ \
//			CUDA_SAFE_CALL( hipDeviceSynchronize() );\
//			CUT_SAFE_CALL(cutStartTimer(timer)); \
//		}
//	#define STOP_TIMER(timer) \
//		if(!param.notimer){ \
//		CUDA_SAFE_CALL( hipDeviceSynchronize() );\
//		CUT_SAFE_CALL(cutStopTimer(timer)); \
//		}

	// timers
//	unsigned int timerTotal, timerUpdateA, timerAfterSVD, timerRT;

	if(!param.notimer)
	{
//		CUT_SAFE_CALL(cutCreateTimer(&timerUpdateA));
//		CUT_SAFE_CALL(cutCreateTimer(&timerAfterSVD));
//        CUT_SAFE_CALL(cutCreateTimer(&timerRT));
	}


//	CUT_SAFE_CALL(cutCreateTimer(&timerTotal));
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
//	CUT_SAFE_CALL(cutStartTimer(timerTotal));

	//
	// initializing cublas
	//
	cublasInit();

	// EM-ICP main loop
	int Titer = 1;
	


	while(sigma_p2 > sigma_inf)
	{
        // fprintf(stderr, "1-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

        Titer++;
//        fprintf(stderr, "%d %f | ", Titer, sigma_p2);
//         fprintf(stderr, "%d iter. sigma_p2 %f  ", Titer++, sigma_p2);
        // fprintf(stderr, "time %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

        #ifndef NOVIEWER
        if(!param.noviewer)
            if (!EngineIteration()) // PointCloudViewer
                break;
        #endif


		//
		// UpdateA
		//

//		START_TIMER(timerUpdateA);

		updateA <<< dimGridForA, dimBlockForA >>>
			(rowsA, colsA, pitchA,
			d_Xx, d_Xy, d_Xz, 
			d_Yx, d_Yy, d_Yz,
			d_R, d_t, 
			d_A, sigma_p2);

//		STOP_TIMER(timerUpdateA);
		
		//
		// Normalization of A
		//

		// hipblasSgemv (char trans, int m, int n, float alpha, const float *A, int lda,
		//              const float *x, int incx, float beta, float *y, int incy)
		//    y = alpha * op(A) * x + beta * y,

		// A * one vector = vector with elements of row-wise sum
		//     d_A      *    d_one    =>  d_C
		//(rowsA*colsA) *  (colsA*1)  =  (rowsA*1)
		hipblasSgemv('n',          // char trans
			  rowsA, colsA, // int m (rows of A), n (cols of A) ; not op(A)
			  1.0f,         // float alpha
			  d_A, pitchA,  // const float *A, int lda
			  d_one, 1,     // const float *x, int incx
			  0.0f,         // float beta
			  d_C, 1);      // float *y, int incy
			  
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
			l_bNoError = false;
            fprintf(stderr,"CUDA ERROR !\n");
		}
//        fprintf(stderr, "2-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
		// void hipblasSaxpy (int n, float alpha, const float *x, int incx, float *y, int incy)
		// alpha * x + y => y
		// exp(-d_0^2/sigma_p2) * d_one + d_C => d_C
		hipblasSaxpy(rowsA, expf(-d_02/sigma_p2), d_one, 1, d_C, 1);

		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
			l_bNoError = false;
            fprintf(stderr,"CUDA ERROR !\n");
		}

		normalizeRowsOfA
			<<< dimGridForA, dimBlockForA >>>
                (rowsA, colsA, pitchA, d_A, d_C);


		//
		// update R,T
		//

		///////////////////////////////////////////////////////////////////////////////////// 

		// compute lambda
      
		// A * one vector = vector with elements of row-wise sum
		//     d_A      *    d_one    =>  d_lambda
		//(rowsA*colsA) *  (colsA*1)  =  (rowsA*1)
		hipblasSgemv('n',          // char trans
			  rowsA, colsA, // int m (rows of A), n (cols of A) ; not op(A)
			  1.0f,         // float alpha
			  d_A, pitchA,  // const float *A, int lda
			  d_one, 1,     // const float *x, int incx
			  0.0f,         // float beta
			  d_lambda, 1); // float *y, int incy
			  
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
			l_bNoError = false;
            fprintf(stderr,"CUDA ERROR !\n");
		}

		// float hipblasSasum (int n, const float *x, int incx) 
		float sumLambda = hipblasSasum (rowsA, d_lambda, 1);
		
		///////////////////////////////////////////////////////////////////////////////////// 

		// compute X'

		// hipblasSgemm (char transa, char transb, int m, int n, int k, float alpha, 
		//              const float *A, int lda, const float *B, int ldb, float beta, 
		//              float *C, int ldc)
		//   C = alpha * op(A) * op(B) + beta * C,
		//
		// m      number of rows of matrix op(A) and rows of matrix C
		// n      number of columns of matrix op(B) and number of columns of C
		// k      number of columns of matrix op(A) and number of rows of op(B) 

		// A * X => X'
		//     d_A      *    d_X    =>  d_Xprime
		//(rowsA*colsA) *  (colsA*3)  =  (rowsA*3)
		//   m  * k           k * n        m * n   
		hipblasSgemm('n', 'n', rowsA, 3, colsA,
			  1.0f, d_A, pitchA,
			  d_X, colsA,
			  0.0f, d_Xprime, rowsA);
			  
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
			l_bNoError = false;
            fprintf(stderr,"CUDA ERROR !\n");
		}

		// X' ./ lambda => X'
		elementwiseDivision
			<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
                (rowsA, d_XprimeX, d_XprimeY, d_XprimeZ, d_lambda);

//        fprintf(stderr, "3-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
		///////////////////////////////////////////////////////////////////////////////////// 

		//
		// centering X' and Y
		//

		///////////////////////////////////////////////////////////////////////////////////// 

		// find weighted center of X' and Y

		// d_Xprime^T *    d_lambda     =>   h_Xc
		//  (3 * rowsA)   (rowsA * 1)  =  (3 * 1)
		hipblasSgemv('t',          // char trans
			  rowsA, 3,     // int m (rows of A), n (cols of A) ; not op(A)
			  1.0f,         // float alpha
			  d_Xprime, rowsA,  // const float *A, int lda
			  d_lambda, 1,     // const float *x, int incx
			  0.0f,         // float beta
			  d_Xc, 1);     // float *y, int incy

//         fprintf(stderr, "31-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
			  
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
            fprintf(stderr,"CUDA ERROR !\n");
			l_bNoError = false;
		}
			
		// d_Y^T *    d_lambda     =>   h_Yc
		//  (3 * rowsA)   (rowsA * 1)  =  (3 * 1)
		hipblasSgemv('t',          // char trans
			  rowsA, 3,     // int m (rows of A), n (cols of A) ; not op(A)
			  1.0f,         // float alpha
			  d_Y, rowsA,  // const float *A, int lda
			  d_lambda, 1,     // const float *x, int incx
			  0.0f,         // float beta
			  d_Yc, 1);     // float *y, int incy
		
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
            fprintf(stderr,"CUDA ERROR !\n");
			l_bNoError = false;
		}
		
//         fprintf(stderr, "32-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

		// void hipblasSscal (int n, float alpha, float *x, int incx)
		// it replaces x[ix + i * incx] with alpha * x[ix + i * incx]
		hipblasSscal (3, 1/sumLambda, d_Xc, 1);

		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
            fprintf(stderr,"CUDA ERROR !\n");
			l_bNoError = false;
		}		
		
		hipblasSscal (3, 1/sumLambda, d_Yc, 1);
		
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
			l_bNoError = false;
		}		

		CUDA_SAFE_CALL(hipMemcpy(h_Xc, d_Xc, sizeof(float)*3, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(h_Yc, d_Yc, sizeof(float)*3, hipMemcpyDeviceToHost));

		///////////////////////////////////////////////////////////////////////////////////// 
//        fprintf(stderr, "4-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
		// centering X and Y

		// d_Xprime .- d_Xc => d_XprimeCenterd
		// d_Y      .- d_Yc => d_YCenterd
		centeringXandY
			<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
				(rowsA, 
				 d_Xc, d_Yc,
				 d_XprimeX, d_XprimeY, d_XprimeZ,
				 d_Yx, d_Yy, d_Yz,
				 d_XprimeCenterdX, d_XprimeCenterdY, d_XprimeCenterdZ,
                 d_YCenterdX, d_YCenterdY, d_YCenterdZ);


		// XprimeCented .* d_lambda => XprimeCented
		elementwiseMultiplication
			<<< blocksPerGridForYsize, threadsPerBlockForYsize>>>
                (rowsA, d_XprimeCenterdX, d_XprimeCenterdY, d_XprimeCenterdZ, d_lambda);

		///////////////////////////////////////////////////////////////////////////////////// 

		// compute S

		//  d_XprimeCented^T *   d_YCenterd     =>  d_S
		//    (3*rowsA)  *  (rowsA*3)  =  (3*3)
		//   m  * k           k * n        m * n
		hipblasSgemm('t', 'n', 3, 3, rowsA,
			  1.0f, d_XprimeCenterd, rowsA,
			  d_YCenterd, rowsA,
			  0.0f, d_S, 3);

			  
		if(cublasGetError() != HIPBLAS_STATUS_SUCCESS)
		{
            fprintf(stderr,"CUDA ERROR !\n");
			l_bNoError = false;
		}
//        fprintf(stderr, "5-- %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);
		CUDA_SAFE_CALL(hipMemcpy(h_S, d_S, sizeof(float)*9, hipMemcpyDeviceToHost));

		///////////////////////////////////////////////////////////////////////////////////// 

		// find RT from S

//		START_TIMER(timerAfterSVD);

		findRTfromS(h_Xc, h_Yc, h_S, h_R, h_t);

//		STOP_TIMER(timerAfterSVD);

		///////////////////////////////////////////////////////////////////////////////////// 

		// copy R,t to device

//		START_TIMER(timerRT);

		CUDA_SAFE_CALL(hipMemcpy(d_R, h_R, sizeof(float)*3*3, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_t, h_t, sizeof(float)*3,   hipMemcpyHostToDevice));

//		STOP_TIMER(timerRT);

		///////////////////////////////////////////////////////////////////////////////////// 

		#ifndef NOVIEWER
		      if(!param.noviewer)
			UpdatePointCloud2(Ysize, param.points2, h_Y, h_R, h_t);
		#endif

		sigma_p2 *= sigma_factor;
//        fprintf(stderr, " %d ", cublasGetError());
	}
//     fprintf(stderr, "End Emicp : ");
//     fprintf(stderr, "%d iter. sigma_p2 %f  ", Titer, sigma_p2);
//     fprintf(stderr, "Final time %.10f [s]\n\n", cutGetTimerValue(timerTotal) / 1000.0f);

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
//    CUT_SAFE_CALL(cutStopTimer(timerTotal));

    // fprintf(stderr, "Emicp computing time: %.10f [s]\n", cutGetTimerValue(timerTotal) / 1000.0f);

	if(!param.notimer)
	{
        //fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerUpdateA)  / 1000.0f, "updateA");
        //fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerAfterSVD) / 1000.0f, "afterSVD");
        //fprintf(stderr, "Average %.10f [s] for %s\n", cutGetAverageTimerValue(timerRT) / 1000.0f, "RT");

//		CUT_SAFE_CALL(cutDeleteTimer(timerTotal));
//		CUT_SAFE_CALL(cutDeleteTimer(timerUpdateA));
//		CUT_SAFE_CALL(cutDeleteTimer(timerAfterSVD));
//		CUT_SAFE_CALL(cutDeleteTimer(timerRT));
	}
	

	cublasShutdown();

	CUDA_SAFE_CALL(hipFree(d_X));
	CUDA_SAFE_CALL(hipFree(d_Y));
	CUDA_SAFE_CALL(hipFree(d_Xprime));
	CUDA_SAFE_CALL(hipFree(d_YCenterd));
	CUDA_SAFE_CALL(hipFree(d_Xc));
	CUDA_SAFE_CALL(hipFree(d_Yc));

	CUDA_SAFE_CALL(hipFree(d_R));
	CUDA_SAFE_CALL(hipFree(d_t));
	CUDA_SAFE_CALL(hipFree(d_A));

	CUDA_SAFE_CALL(hipFree(d_S));
	CUDA_SAFE_CALL(hipFree(d_one));
	CUDA_SAFE_CALL(hipFree(d_sumOfMRow));
	CUDA_SAFE_CALL(hipFree(d_C));
	CUDA_SAFE_CALL(hipFree(d_lambda));

	// CUDA_SAFE_CALL( hipDeviceReset() ); // depreciated / moved to cudaReset
	
    if(hipGetLastError() != HIPBLAS_STATUS_SUCCESS)
	{
        fprintf(stderr,"CUDA ERROR !\n");
		l_bNoError = false;
	}
	
  
	delete [] h_one;
	
	return l_bNoError;
}


//// TEST CUDA FUNCTIONS

#define LINEAR_BLOCK_SIZE 512
#define IDC2D(i, j, ld)(((j)*(ld))+(i))

// Gauss-Jordan elimination row switching kernel code.
__global__ void switchRows( float *matrix,float *result, int index, int rowToSwitch, int lda )
{
    int y = threadIdx.y + LINEAR_BLOCK_SIZE * blockIdx.y;
    float tmp_m, tmp_r;
    if ( y < lda )
    {
        tmp_m = matrix[ IDC2D( index, y, lda ) ];
        matrix[ IDC2D( index, y, lda ) ] = matrix[ IDC2D( rowToSwitch, y, lda ) ];
        matrix[ IDC2D( rowToSwitch, y, lda ) ] = tmp_m;
        tmp_r = result[ IDC2D( index, y, lda ) ];
        result[ IDC2D( index, y, lda ) ] = result[ IDC2D( rowToSwitch, y, lda ) ];
        result[ IDC2D( rowToSwitch, y, lda ) ] = tmp_r;
    }
}

// Gauss-Jordan elimination pivot row normalization kernel code
__global__ void normalizePivotRow( float *matrix, float *result, int index, int lda )
{
    // Position of each thread inside the block
    int ty = threadIdx.y;
    // Position of each thread inside the matrix
    int y = ty + LINEAR_BLOCK_SIZE * blockIdx.y;
    // Pivot value of the row
    __shared__ float pivotValue;
    if ( y < lda )
    {
        if ( ty == 0 )
        // First thread of each block loads pivotValue
            pivotValue = matrix[ IDC2D( index, index, lda) ];
        __syncthreads();

        // Every thread divides the element of its position by pivotValue
        matrix[ IDC2D( index, y, lda )] /= pivotValue;
        result[ IDC2D( index, y, lda )] /= pivotValue;
    }
}

// Gauss-Jordan elimination zero-maker kernel code.
__global__ void linearMge( float *matrix, float *result, int index, int lda )
{
    int ty = threadIdx.y;
    int x = blockIdx.x;
    int y = ty + blockIdx.y * LINEAR_BLOCK_SIZE;
    __shared__ float multColumn[ LINEAR_BLOCK_SIZE ];
    __shared__ float matrixPivotValue;
    __shared__ float matrixRow[ LINEAR_BLOCK_SIZE ];
    __shared__ float resultPivotValue;
    __shared__ float resultRow[ LINEAR_BLOCK_SIZE];
    float newMatrixValue; float newResultValue;
    if ( y < lda )
    {
        // Each block loads the value of the pivot Row to be substracted
        if ( ty == 0 )
        {
            matrixPivotValue = matrix[ IDC2D( index, x, lda )];
            resultPivotValue = result[ IDC2D( index, x, lda )];
        }
        multColumn[ ty ] = matrix[ IDC2D( y, index, lda )];
        matrixRow[ ty ] = matrix[ IDC2D( y, x, lda )];
        resultRow[ ty ] = result[ IDC2D( y, x, lda )];
        __syncthreads();

        if ( y!= index )
        {
            newMatrixValue = matrixRow[ty] - multColumn[ty] * matrixPivotValue;
            newResultValue = resultRow[ty] - multColumn[ty] * resultPivotValue;
            // Copy to the matrix
            matrix[ IDC2D( y, x, lda) ] = newMatrixValue;
            result[ IDC2D( y, x, lda) ] = newResultValue;
        }
    }
}
